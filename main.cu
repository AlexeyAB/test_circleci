
#include <hip/hip_runtime.h>
#include <stdio.h>	// printf(); gets();
#include <stdlib.h>	// malloc();
#include <math.h>	// sqrt();

__global__	// execute on GPU
void calculation(int size, float *y) 
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size) y[i] = sqrt( y[i] );
}

// execute on CPU
int main(void) {
	int size = 1000000;		// 1M elements
	float *array_cpu_ptr;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&array_cpu_ptr, size*sizeof(float), 
			hipHostMallocMapped);
	float *array_ptr;		// UVA pointer
	hipHostGetDevicePointer(&array_ptr, array_cpu_ptr, 0);

	// Init array
	for (int i = 0; i < size; i++) array_ptr[i] = 2.0f;

	int blocks_num = size/256 + 1;
	int threads_num = 256;

	// execute on GPU
	calculation<<<blocks_num, threads_num>>>(size, array_ptr);

	// CPU waiting for completion of GPU execution
	hipDeviceSynchronize(); 
	printf("array_ptr[10]: %f \n", array_ptr[10]);
	hipHostFree(array_cpu_ptr);
	hipDeviceReset();		// reset current device
	return 0;
}
